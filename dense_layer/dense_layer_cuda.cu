#include "hip/hip_runtime.h"
/**
 * Implementation of a Dense (Tensorflow) or Fully Connected (PyTorch) network layer
 *
 * @author: Yvo Elling
 * @date: 10-03-23
 */

#include <iostream>
#include <cstdlib>
#include <cstdint>
#include <array>
#include <chrono>

#include <stdio.h>
#include "hw_data.h"

typedef uint8_t CoreIdx;

#define VECTOR_LENGTH 1'000'000
#define NROF_TEST_RUNS 100

using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::chrono::milliseconds;

__global__ void computeDenseLayerCUDA(float* weights, float* input, float* bias, float* output) {
    CoreIdx idx = threadIdx.x;
    
    float nodeOutputSum = 0.0f;
    for (int i = 0; i < VECTOR_LENGTH; ++i) {
        nodeOutputSum += input[idx] * weights[i] + bias[i];
    }
    output[idx] = nodeOutputSum;
}

void computeDenseLayerCPU(float* weights, float* input, float* bias, float* output) {
    float nodeOutputSum = 0.0f;
    auto t1 = high_resolution_clock::now();
    for (int idx = 0; idx < VECTOR_LENGTH; ++idx) {
        for (int i = 0; i < VECTOR_LENGTH; ++i) {
            nodeOutputSum += input[idx] * weights[i] + bias[i];
        }
        output[idx] = nodeOutputSum;
    }
    auto t2 = high_resolution_clock::now();
    int total_execution_time = duration_cast<milliseconds>(t2 - t1).count();
    std::cout << "Total execution time on CPU is: " << total_execution_time << " ms" << std::endl;
}

int main (int argc, char** argv) {
    std::cout << "Starting CUDA Application" << std::endl;
    std::cout << "Launching CUDA Program for Dense Layer" << std::endl;

    auto h_weights = (float *)calloc(VECTOR_LENGTH, sizeof(float));
    auto h_input = (float *)calloc(VECTOR_LENGTH, sizeof(float));
    auto h_bias = (float *)calloc(VECTOR_LENGTH, sizeof(float));
    auto h_output = (float *)calloc(VECTOR_LENGTH, sizeof(float));

    float *d_weights, *d_input, *d_bias, *d_output;

    hipMalloc((void**)&d_weights, VECTOR_LENGTH * sizeof(float));
    hipMalloc((void**)&d_input, VECTOR_LENGTH * sizeof(float));
    hipMalloc((void**)&d_bias, VECTOR_LENGTH * sizeof(float));
    hipMalloc((void**)&d_output, VECTOR_LENGTH * sizeof(float));

    hipMemcpy(d_weights, h_weights, VECTOR_LENGTH*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input, h_input, VECTOR_LENGTH*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, h_bias, VECTOR_LENGTH*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_output, h_output, VECTOR_LENGTH*sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    std::array<float, NROF_TEST_RUNS> execution_times;

    for (int i = 0; i < NROF_TEST_RUNS; ++i) {
        hipEventRecord(start);
        computeDenseLayerCUDA<<<QUADRO_P2000_SM*3, QUADRO_P200_THREADS_PER_SM*3>>>(d_weights, d_input, d_bias, d_output);
        hipEventRecord(stop);
    
        hipDeviceSynchronize();
        hipMemcpy(h_output, d_output, VECTOR_LENGTH*sizeof(float), hipMemcpyDeviceToHost);
    
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        execution_times[i] = milliseconds;
    }

    float execution_time_sum = 0;
    for (int i = 0; i < NROF_TEST_RUNS; ++i) {
        execution_time_sum += execution_times[i];
    }
    float avg_execution_time = execution_time_sum / execution_times.size();
    std::cout << "Total average kernel execution time is: " << avg_execution_time << "ms" << std::endl;

    computeDenseLayerCPU(h_weights, h_input, h_bias, h_output);

    hipFree(d_weights);
    hipFree(d_input);
    hipFree(d_bias);
    hipFree(d_output);

    free(h_weights);
    free(h_input);
    free(h_bias);
    free(h_output);
}